#include "hip/hip_runtime.h"
#include <torch/types.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <stdio.h>

#define BLOCK_M 32
#define BLOCK_N 32

__device__ inline float load_or(const float* ptr, int idx, int max_idx, float other) {
    if (idx < 0 || idx >= max_idx) {
        return other;
    }
    return ptr[idx];
}

__global__
void sparse_attention_forward_kernel(
    // Q, K, V
    const float* __restrict__ Q,
    const float* __restrict__ K,
    const float* __restrict__ V,
    // Q_idx, K_idx
    const int* __restrict__ Q_idx,
    const int* __restrict__ K_idx,
    // for storing partial sums
    float* __restrict__ L,
    float* __restrict__ M,
    // final output
    float* __restrict__ Out,
    // metadata
    const float softmax_scale,
    int B,
    int H,
    int NQ,
    int NK,
    int d
)
{
    int block_m_id = blockIdx.x;  
    int bh_id = blockIdx.y;
    int thread_id = threadIdx.x;

    int b = bh_id / H;  // batch index
    int h = bh_id % H;  // head index

    size_t bh_offset_qkv = (size_t)b * H * NQ * d + (size_t)h * NQ * d;
    size_t bh_offset_LM = (size_t)(b * H + h) * NQ;

    int start_m = block_m_id * BLOCK_M;
    int q_i   = start_m + thread_id;  // local index for the queries

    size_t q_offset = bh_offset_qkv + (size_t)q_i * d;

    float m_prev = -INFINITY;
    float l_prev = 0.0f;
    __shared__ float acc_s[BLOCK_M * 128];

    float q_val[128];  // for d <= 128
    #pragma unroll
    for (int dim = 0; dim < d; dim++) {
        q_val[dim] = 0.0f;
    }

    // "masked" loads
    if (q_i < NQ) {
        #pragma unroll
        for (int dim = 0; dim < d; dim++) {
            q_val[dim] = Q[q_offset + dim];
        }
        l_prev = L[bh_offset_LM + q_i]; 
        m_prev = M[bh_offset_LM + q_i]; 
    }

    float acc[128];
    #pragma unroll
    for (int dim = 0; dim < d; dim++) {
        acc[dim] = 0.0f;
    }

    int Qi = -1;
    if (q_i < NQ) {
        size_t bh_offset_idxQ = (size_t)b * H * NQ + (size_t)h * NQ;
        Qi = Q_idx[bh_offset_idxQ + q_i];
    }

    int max_Qi_block = Qi;

    __shared__ int qi_sh[BLOCK_M];
    if (thread_id < BLOCK_M) {
        qi_sh[thread_id] = (q_i < NQ) ? Qi : -1; 
    }
    __syncthreads();

    if (thread_id == 0) {
        int block_max = -1;
        for (int i = 0; i < BLOCK_M; i++) {
            block_max = max(block_max, qi_sh[i]);
        }
        qi_sh[0] = block_max;
    }
    __syncthreads();
    int block_max_Qi = qi_sh[0];

    int end_n_blocks = 0;
    for (int start_n = 0; start_n < NK; start_n += BLOCK_N) {
        float local_min = 1e9f;
        for (int x = thread_id; x < BLOCK_N; x += blockDim.x) {
            int kn = start_n + x;
            int val = (kn < NK)
                ? K_idx[(b * H + h) * NK + kn]
                : 1e9; 
            if (val < local_min) {
                local_min = (float)val;
            }
        }
        // reduce among threads
        __shared__ float min_sh[BLOCK_M];
        min_sh[thread_id] = local_min;
        __syncthreads();

        // block-wide reduce min
        if (thread_id == 0) {
            float block_min = 1e9f;
            for (int i = 0; i < BLOCK_M; i++) {
                if (min_sh[i] < block_min) {
                    block_min = min_sh[i];
                }
            }
            min_sh[0] = block_min;
        }
        __syncthreads();
        float block_min_ki = min_sh[0];

        // If the min_ki <= max_Qi, we need that block
        // If block_min_ki == 1e9, that means it's out of range
        if (block_min_ki <= block_max_Qi && block_min_ki < 1e9) {
            end_n_blocks += 1;
        } else {
            // no more blocks needed
        }
    }

    int n_blocks_done = 0;
    for (int start_n = 0; start_n < NK && n_blocks_done < end_n_blocks; start_n += BLOCK_N) {
        float local_min = 1e9f;
        for (int x = thread_id; x < BLOCK_N; x += blockDim.x) {
            int kn = start_n + x;
            int val = (kn < NK)
                ? K_idx[(b * H + h) * NK + kn]
                : 1e9; 
            if (val < local_min) {
                local_min = (float)val;
            }
        }
        __shared__ float min_sh2[BLOCK_M];
        min_sh2[thread_id] = local_min;
        __syncthreads();
        if (thread_id == 0) {
            float block_min_ki = 1e9f;
            for (int i = 0; i < BLOCK_M; i++) {
                block_min_ki = fminf(block_min_ki, min_sh2[i]);
            }
            min_sh2[0] = block_min_ki;
        }
        __syncthreads();
        float block_min_ki = min_sh2[0];
        // check if we skip
        if (block_min_ki > block_max_Qi || block_min_ki >= 1e9f) {
            // skip
            continue;
        }
        n_blocks_done += 1;

        float qk[BLOCK_N];
        #pragma unroll
        for (int i = 0; i < BLOCK_N; i++) {
            qk[i] = -INFINITY;  
        }

        __shared__ float k_sh[BLOCK_N * 128];
        __shared__ float v_sh[BLOCK_N * 128];
        __shared__ int   ki_sh[BLOCK_N];

        for (int x = thread_id; x < BLOCK_N; x += blockDim.x) {
            int kn = start_n + x;
            if (kn < NK) {
                ki_sh[x] = K_idx[(b * H + h) * NK + kn];
            } else {
                ki_sh[x] = 1e9;
            }
        }

        for (int x = thread_id; x < BLOCK_N * d; x += blockDim.x) {
            int block_k = x / d;  // 0..BLOCK_N-1
            int dim_k   = x % d;
            int kn = start_n + block_k;
            float kval = 0.0f;
            float vval = 0.0f;
            if (kn < NK) {
                size_t offset_k = bh_offset_qkv + (size_t)kn * d;
                kval = K[offset_k + dim_k];
                vval = V[offset_k + dim_k];
            }
            k_sh[x] = kval;
            v_sh[x] = vval;
        }
        __syncthreads();

        if (q_i < NQ) {
            for (int j = 0; j < BLOCK_N; j++) {
                // check if j < NK
                int kn = start_n + j;
                if (kn >= NK) {
                    break; // out of range
                }
                // causal mask: if Qi < Ki => -inf
                if (Qi >= ki_sh[j]) {
                    // do dot
                    float sum = 0.f;
                    #pragma unroll
                    for (int dim_k = 0; dim_k < d; dim_k++) {
                        sum += q_val[dim_k] * k_sh[j * d + dim_k];
                    }
                    sum *= softmax_scale;
                    qk[j] = sum;
                } else {
                    qk[j] = -INFINITY;
                }
            }
        }
        __syncthreads();

        float local_max = -INFINITY;
        #pragma unroll
        for (int j = 0; j < BLOCK_N; j++) {
            float val = qk[j];
            if (val > local_max) {
                local_max = val;
            }
        }
        float m_curr = fmaxf(local_max, m_prev);

        // compute p = exp(qk - m_curr), scaled old terms
        float sum_p = 0.f;
        #pragma unroll
        for (int j = 0; j < BLOCK_N; j++) {
            if (qk[j] > -INFINITY) {
                float val = expf(qk[j] - m_curr);
                qk[j] = val;  // reuse
                sum_p += val;
            } else {
                qk[j] = 0.f;
            }
        }
        // combine with old l
        float l_curr = sum_p + l_prev * expf(m_prev - m_curr);

        if (q_i < NQ && l_curr > 0.f) {
            float scale_old = (l_prev == 0.f) ? 0.f : (l_prev * expf(m_prev - m_curr)) / l_curr;
            float scale_new = 1.f / l_curr;
            #pragma unroll
            for (int dim_k = 0; dim_k < d; dim_k++) {
                acc[dim_k] = acc[dim_k] * scale_old; 
            }
            // Now accumulate p * V
            for (int j = 0; j < BLOCK_N; j++) {
                float p_ij = qk[j] * scale_new;
                // add dot with v_sh
                #pragma unroll
                for (int dim_k = 0; dim_k < d; dim_k++) {
                    acc[dim_k] += p_ij * v_sh[j * d + dim_k];
                }
            }
            // update l_prev, m_prev
            l_prev = l_curr;
            m_prev = m_curr;
        }
        __syncthreads();
    } // end loop over blocks in K

    if (q_i < NQ) {
        L[bh_offset_LM + q_i] = l_prev;
        M[bh_offset_LM + q_i] = m_prev;
        size_t out_offset = bh_offset_qkv + (size_t)q_i * d;
        for (int dim_k = 0; dim_k < d; dim_k++) {
            Out[out_offset + dim_k] = acc[dim_k];
        }
    }
}


torch::Tensor forward(
    torch::Tensor Q,
    torch::Tensor K,
    torch::Tensor V,
    torch::Tensor Q_idx,
    torch::Tensor K_idx,
    float sm_scale
) {
    // Expect shapes [B,H,NQ,d], [B,H,NK,d], ...
    int B = Q.size(0);
    int H = Q.size(1);
    int NQ = Q.size(2);
    int d  = Q.size(3);
    int NK = K.size(2);

    auto Out = torch::zeros_like(Q);
    auto L = torch::zeros({B, H, NQ}, Q.options().dtype(torch::kFloat32));
    auto M = torch::full({B, H, NQ}, -INFINITY, Q.options().dtype(torch::kFloat32));

    dim3 block_dim(BLOCK_M);
    dim3 grid_dim(
        (NQ + BLOCK_M - 1) / BLOCK_M, // # blocks covering all queries
        B * H                         // batch*head
    );

    sparse_attention_forward_kernel<<<grid_dim, block_dim>>>(
        Q.data_ptr<float>(),
        K.data_ptr<float>(),
        V.data_ptr<float>(),
        Q_idx.data_ptr<int>(),
        K_idx.data_ptr<int>(),
        L.data_ptr<float>(),
        M.data_ptr<float>(),
        Out.data_ptr<float>(),
        sm_scale,
        B, H, NQ, NK, d
    );
    return Out;
}
